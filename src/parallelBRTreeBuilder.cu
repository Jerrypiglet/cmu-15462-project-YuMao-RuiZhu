#include "hip/hip_runtime.h"
#include "parallelBRTreeBuilder.h"
#include <iostream>

#define DEFAULT_THREAD_PER_BLOCK 1024

/*check error code of hipMalloc and print out if needed*/
#define safe_cuda(CODE)\
 {\
  hipError_t err = CODE;\
  if(err != hipSuccess) {\
    std::cout<<"CUDA error:"<<hipGetErrorString(err)<<std::endl;\
 }\
}

/**
 * alloc a memory on gpu and copy data from cpu to gpu.
 */
inline void copyFromCPUtoGPU(void** dst, void* src, int size)
{
   hipMalloc(dst, size);
   safe_cuda(hipMemcpy(*dst, src, size, hipMemcpyHostToDevice));
}

/**
 * alloc a memory on cpu and copy data from gpu to cpu.
 */
inline void copyFromGPUtoCPU(void** dst, void* src, int size)
{
   *dst = malloc(size);
   safe_cuda(hipMemcpy(*dst, src, size, hipMemcpyDeviceToHost));
}

/**
 * intialize parallelBRTreeBuilder by copying the data needed
 * from host memory (CPU) to device memory (GPU), initialize
 * data members such as configuration parameters.
 */
ParallelBRTreeBuilder::ParallelBRTreeBuilder(unsigned int* const sorted_morton_code, BBox* const bboxes, int size):
 d_sorted_morton_code(0),
 d_leaf_nodes(0),
 h_leaf_nodes(0),
 d_internal_nodes(0),
 h_internal_nodes(0), 
 numInternalNode(size-1),
 numLeafNode(size)
{
   //copy data from cpu to gpu
   copyFromCPUtoGPU((void**)&d_sorted_morton_code, sorted_morton_code, size * sizeof(unsigned int));
   copyFromCPUtoGPU((void**)&d_bboxes, bboxes, size * sizeof(BBox));
   
   //initialize d_leaf_nodes and d_internal_nodes
   h_leaf_nodes = (BRTreeNode*)calloc(numLeafNode, sizeof(BRTreeNode));
   for (int idx = 0; idx < numLeafNode; idx++) { 
      h_leaf_nodes[idx].setIdx(idx);
      h_leaf_nodes[idx].bbox = BBox();
   }
   copyFromCPUtoGPU((void**)&d_leaf_nodes,h_leaf_nodes,numLeafNode * sizeof(BRTreeNode));
   free(h_leaf_nodes);

   h_internal_nodes = (BRTreeNode*)calloc(numInternalNode, sizeof(BRTreeNode));
   for (int idx = 0; idx < numInternalNode; idx++) { 
      h_internal_nodes[idx].setIdx(idx);
      h_internal_nodes[idx].bbox = BBox();
   }
   copyFromCPUtoGPU((void**)&d_internal_nodes,h_internal_nodes,numInternalNode * sizeof(BRTreeNode));
   free(h_internal_nodes);
}

/**
 * delta operator measures the common prefix of two morton_code
 * if j is not in the range of the sorted_morton_code,
 * delta operator returns -1.
 */
__device__ int delta(int i, int j, unsigned int* sorted_morton_code, int length)
{
  if(j<0||j>=length)
  {
    return -1;
  }
  else
  {
    return __clz(sorted_morton_code[i] ^ sorted_morton_code[j]);
  }
}

/**
 * determine the range of an internal node
 */
__device__ int2 determineRange(unsigned int* sorted_morton_code, int numInternalNode, int i)
{
  int size = numInternalNode+1;
  int d = delta(i, i+1, sorted_morton_code, size) - delta(i, i-1, sorted_morton_code, size);
  d = d > 0? 1:-1;
  
  //compute the upper bound for the length of the range
  int delta_min = delta(i,i-d,sorted_morton_code, size);
  int lmax = 2;
  while(delta(i,i+lmax*d,sorted_morton_code,size)>delta_min)
  {
    lmax = lmax * 2;
  }
  
  //find the other end using binary search
  int l=0;
  for(int t = lmax/2; t>=1 ;t/=2)
  {
    if(delta(i,i+(l+t)*d,sorted_morton_code,size)>delta_min)
    {
      l= l+t;
    }
  }
  int j = i+l*d;
  
  int2 range;
  if(i<=j) { range.x = i; range.y = j; }
  else     { range.x = j; range.y = i; }
  return range;
}

/**
 * to judge if two values differes 
 * in bit position n
 */
__device__ bool is_diff_at_bit(unsigned int val1, unsigned int val2, int n)
{
  return val1>>(31-n) != val2>>(31-n);
}

/**
 * find the best split position for an internal node
 */
__device__ int findSplit(unsigned int* sorted_morton_code, int start, int last)
{
  //return -1 if there is only 
  //one primitive under this node.
  if(start == last) 
  {
    return -1;
  }
  else
  {
    int common_prefix = __clz(sorted_morton_code[start] ^ sorted_morton_code[last]);

    //handle duplicated morton code separately
    if(common_prefix == 32)
    {
      return (start + last)/2;
    }

    // Use binary search to find where the next bit differs.
    // Specifically, we are looking for the highest object that
    // shares more than commonPrefix bits with the first one.

    int split = start; // initial guess
    int step = last - start;
    do
    {
        step = (step + 1) >> 1; // exponential decrease
        int newSplit = split + step; // proposed new position

        if (newSplit < last)
        {
            bool is_diff = is_diff_at_bit(sorted_morton_code[start],
                                          sorted_morton_code[newSplit],
                                          common_prefix);
            if(!is_diff)
            {
              split = newSplit; // accept proposal
            }
        }
    }
    while (step > 1);

    return split;
  }
}

//FOR BR-TREE CONSTRUCTION
//TODO: implement internal node processing routine
//TODO: handle duplicated morton codes as special case (using their position i,j as fallback)

//FOR BVH CONSTRUCTION
//TODO: implement AABB construction process by go back from the tree node to the root
//TODO: convert BR-TREE BACK TO BVH
//TODO: debug
__global__ static void processInternalNode(unsigned int* sorted_morton_code, int numInternalNode,
					   BRTreeNode* leafNodes,
				           BRTreeNode* internalNodes)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= numInternalNode ) return;
  
  // Find out which range of objects the node corresponds to.
  int2 range = determineRange(sorted_morton_code, numInternalNode, idx);
  int first = range.x;
  int last = range.y;

  // Determine where to split the range.
  int split = findSplit(sorted_morton_code, first, last);

  if(split == -1) return;

  // Select childA.
  BRTreeNode* childA;
  bool isChildALeaf = false;
  if (split == first) {
      childA = &(leafNodes[split]);
      isChildALeaf = true;
  } else childA = &(internalNodes[split]);

  // Select childB.
  BRTreeNode* childB;
  bool isChildBLeaf = false;
  if (split + 1 == last) {
      childB = &(leafNodes[split + 1]);
      isChildBLeaf = true;
  }
  else childB = &(internalNodes[split + 1]);

  // Record parent-child relationships.
  internalNodes[idx].setChildA(split,isChildALeaf);
  internalNodes[idx].setChildB(split+1,isChildBLeaf);
  childA->setParent(idx);
  childB->setParent(idx);
}

/**
 * construct bounding boxes from leaf up to root
 */
__global__ static void calculateBoudingBox(BBox* d_bboxes, int numLeafNode,
			                   BRTreeNode* leafNodes, BRTreeNode* internalNodes)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= numLeafNode ) return;
  
  //handle leaf first
  BRTreeNode* node = &leafNodes[idx];
  node->bbox = d_bboxes[idx];

  //terminate if it is root node
  bool is_null = false;
  int parentIdx = node->getParent(is_null);
  if(is_null) return; 
  node = &internalNodes[parentIdx];

  int initial_val = atomicInc(&node->counter,1);  
  while(1)
  {
    if(initial_val == 0) return; //terminate the first accesing thread
    
    //calculate bounding box by merging two children's bounding box
    bool is_leaf = false;  
    int childAIdx = node->getChildA(is_leaf, is_null);
    if(is_leaf) node->bbox.expand(leafNodes[childAIdx].bbox);
    else node->bbox.expand(internalNodes[childAIdx].bbox);

    int childBIdx = node->getChildB(is_leaf, is_null);
    if(is_leaf) node->bbox.expand(leafNodes[childBIdx].bbox);
    else node->bbox.expand(internalNodes[childBIdx].bbox); 
    
    //terminate if it is root node
    parentIdx = node->getParent(is_null);
    if(is_null) return; 
    node = &internalNodes[parentIdx];   
    initial_val = atomicInc(&node->counter,1);
  } 
}

/**
 * build binary radix tree on GPU
 */
void ParallelBRTreeBuilder::build()
{
  //build the bvh
  int threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
  int numBlock = (numInternalNode+DEFAULT_THREAD_PER_BLOCK-1)/threadPerBlock; 
  processInternalNode<<<numBlock,threadPerBlock>>>(d_sorted_morton_code, numInternalNode,
						   d_leaf_nodes, d_internal_nodes);
  
  //calculate bounding box
  threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
  numBlock = (numLeafNode+DEFAULT_THREAD_PER_BLOCK-1)/threadPerBlock; 
  calculateBoudingBox<<<numBlock,threadPerBlock>>>(d_bboxes, numLeafNode, 
						   d_leaf_nodes, d_internal_nodes);
}

/**
 * get leaf nodes (host)
 */
BRTreeNode* ParallelBRTreeBuilder::get_leaf_nodes()
{
  copyFromGPUtoCPU((void**)&h_leaf_nodes, d_leaf_nodes, numLeafNode * sizeof(BRTreeNode));   
  return h_leaf_nodes;
}

/**
 * get internal nodes (host)
 */
BRTreeNode* ParallelBRTreeBuilder::get_internal_nodes()
{
  copyFromGPUtoCPU((void**)&h_internal_nodes, d_internal_nodes, numInternalNode * sizeof(BRTreeNode));   
  return h_internal_nodes;
}

/**
 * free memory on host
 */
void ParallelBRTreeBuilder::freeHostMemory()
{
   free(h_leaf_nodes);
   free(h_internal_nodes);
}

/**
 * free memory on device
 */
void ParallelBRTreeBuilder::freeDeviceMemory()
{
   hipFree(d_leaf_nodes);
   hipFree(d_internal_nodes);
   hipFree(d_sorted_morton_code);
}

/**
 * deconstructor
 */
ParallelBRTreeBuilder::~ParallelBRTreeBuilder() {}



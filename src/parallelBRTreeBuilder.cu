#include "hip/hip_runtime.h"
#include "parallelBRTreeBuilder.h"
#include <thrust/sequence.h>
#include <iostream>

#define DEFAULT_THREAD_PER_BLOCK 256

/**
 * intialize parallelBRTreeBuilder by copying the data needed
 * from host memory (CPU) to device memory (GPU), initialize
 * data members such as configuration parameters.
 */
ParallelBRTreeBuilder::ParallelBRTreeBuilder(unsigned int* const sorted_morton_code, int size):
numInternalNode(size-1),
numLeafNode(size)
{
   //copy data from cpu to gpu
   hipMalloc(&d_sorted_morton_code, size * sizeof(unsigned int));
   hipMemcpy(d_sorted_morton_code, sorted_morton_code, size * sizeof(unsigned int), hipMemcpyHostToDevice);
   
   //resize leaf arrays and internal node arrays
   d_leaf_nodes.resize(numLeafNode);
   d_internal_nodes.resize(numInternalNode);
   
   //set configuration parameter for CUDA
   threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
   numBlock = (numInternalNode+DEFAULT_THREAD_PER_BLOCK-1)/threadPerBlock; 
   
   std::cout<<std::endl;
   std::cout<<"-threads per block:"<<threadPerBlock<<std::endl;
   std::cout<<"-number of blocks:"<<numBlock<<std::endl;
   
}


//FOR BR-TREE CONSTRUCTION
//TODO: implement internal node processing routine
//TODO: handle duplicated morton codes as special case (using their position i,j as fallback)

//FOR BVH CONSTRUCTION
//TODO: implement AABB construction process by go back from the tree node to the root
//TODO: convert BR-TREE BACK TO BVH
//TODO: debug
__global__ static void processInternalNode(unsigned int* sorted_morton_code, int numInternalNode)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= numInternalNode ) return;
  
  
}

/**
 * build binary radix tree on GPU
 */
void ParallelBRTreeBuilder::build()
{
  processInternalNode<<<numBlock,threadPerBlock>>>(d_sorted_morton_code, numInternalNode);
}

/**
 * get the leaf nodes (host)
 */
thrust::device_vector<BRTreeNode>* ParallelBRTreeBuilder::get_leaf_nodes()
{
  h_leaf_nodes.resize(numLeafNode);
  thrust::copy(d_leaf_nodes.begin(), d_leaf_nodes.end(), h_leaf_nodes.begin());   
  return &h_leaf_nodes;
}

/**
 * get the internal nodes (host)
 */
thrust::device_vector<BRTreeNode>* ParallelBRTreeBuilder::get_internal_nodes()
{
  h_internal_nodes.resize(numInternalNode);
  thrust::copy(d_internal_nodes.begin(), d_internal_nodes.end(), h_internal_nodes.begin());   
  return &h_internal_nodes;
}

/**
 * deconstructor
 */
ParallelBRTreeBuilder::~ParallelBRTreeBuilder()
{
  //TODO: free stuffs here
}


